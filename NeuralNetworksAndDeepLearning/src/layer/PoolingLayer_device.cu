
#include <hip/hip_runtime.h>
/*
 * PoolingLayer.cpp
 *
 *  Created on: 2016. 5. 23.
 *      Author: jhkim
 */


#ifdef GPU_MODE

#include "PoolingLayer.h"

#define POOLINGLAYER_LOG 0

using namespace std;

template <typename Dtype>
void PoolingLayer<Dtype>::reshape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches 	= inputShape[0];
	uint32_t channels 	= inputShape[1];
	uint32_t rows 		= inputShape[2];
	uint32_t cols 		= inputShape[3];

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			batches, channels, rows, cols));

	//int n = 0, c = 0, h = 0, w = 0;
	/*
	checkCUDNN(cudnnGetPooling2dForwardOutputDim(
			this->pooling_fn->getPoolDesc(),
			this->inputTensorDesc,
			&n, &c, &h, &w));
			*/

	int pooledHeight = static_cast<int>(ceil(static_cast<float>(
			rows + 2 * this->pool_d.pad - this->pool_d.rows) / this->pool_d.stride)) + 1;
	int pooledWidth = static_cast<int>(ceil(static_cast<float>(
			cols + 2 * this->pool_d.pad - this->pool_d.cols) / this->pool_d.stride)) + 1;

	if (this->pool_d.pad) {
		if ((pooledHeight - 1) * this->pool_d.stride >= rows + this->pool_d.pad) {
			pooledHeight--;
		}
		if ((pooledWidth - 1) * this->pool_d.stride >= cols + this->pool_d.pad) {
			pooledWidth--;
		}
		assert((pooledHeight - 1) * this->pool_d.stride < rows + this->pool_d.pad);
		assert((pooledWidth - 1) * this->pool_d.stride < cols + this->pool_d.pad);
	}

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			batches, channels, pooledHeight, pooledWidth));

	/*
	uint32_t obatches = static_cast<uint32_t>(batches);
	uint32_t ochannels = static_cast<uint32_t>(channels);
	uint32_t orows = static_cast<uint32_t>(pooledHeight);
	uint32_t ocols = static_cast<uint32_t>(pooledWidth);
	*/

#if POOLINGLAYER_LOG
	printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
			this->name.c_str(), obatches, ochannels, orows, ocols);
#endif

	this->_inputShape[0] = inputShape;
	this->_outputData[0]->reshape({
		static_cast<uint32_t>(batches),
		static_cast<uint32_t>(channels),
		static_cast<uint32_t>(pooledHeight),
		static_cast<uint32_t>(pooledWidth)});

	/*
	this->setInDimension(this->_inputData[0]->getShape());

	cudnnTensorDescriptor_t tempInputTensorDesc;
	checkCUDNN(cudnnCreateTensorDescriptor(&tempInputTensorDesc));
	checkCUDNN(cudnnSetTensor4dDescriptor(tempInputTensorDesc,
				CUDNN_TENSOR_NCHW,
				CUDNN_DATA_FLOAT,
				this->in_dim.batches, this->in_dim.channels, this->in_dim.rows,
                this->in_dim.cols));

	int n, c, h, w;
	checkCUDNN(cudnnGetPooling2dForwardOutputDim(pooling_fn->getPoolDesc(),
			tempInputTensorDesc,
			&n, &c, &h, &w));

	this->out_dim.batches = n;
	this->out_dim.channels = c;
	this->out_dim.rows = h;
	this->out_dim.cols = w;

	checkCUDNN(cudnnDestroyTensorDescriptor(tempInputTensorDesc));

	if(recursive) {
		Layer<Dtype>::_shape();
	}
	*/
}

template <typename Dtype>
void PoolingLayer<Dtype>::feedforward() {
	reshape();

	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	this->_inputData[0]->print_data();

	this->pooling_fn->forward(this->inputTensorDesc, d_inputData,
			this->outputTensorDesc, d_outputData);

	this->_outputData[0]->print_data();
}

template <typename Dtype>
void PoolingLayer<Dtype>::backpropagation() {
	if (this->_propDown[0]) {
		this->_outputData[0]->print_data();
		this->_inputData[0]->print_data();

		const Dtype* d_outputData = this->_outputData[0]->device_data();
		const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
		const Dtype* d_inputData = this->_inputData[0]->device_data();
		Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();
		this->pooling_fn->backward(this->outputTensorDesc, d_outputData, d_outputGrad,
				this->inputTensorDesc, d_inputData, d_inputGrad);

		this->_inputData[0]->print_grad();
	}
}


template void PoolingLayer<float>::reshape();
template void PoolingLayer<float>::feedforward();
template void PoolingLayer<float>::backpropagation();

#endif
