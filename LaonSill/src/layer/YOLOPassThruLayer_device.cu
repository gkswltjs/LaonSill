#include "hip/hip_runtime.h"
/**
 * @file YOLOPassThruLayer_device.cu
 * @date 2018-01-03
 * @author moonhoen lee
 * @brief 
 * @details
 */

#include "YOLOPassThruLayer.h"
#include "PropMgmt.h"
#include "MemoryMgmt.h"

using namespace std;

#define EPSILON                 0.000001

template <typename Dtype>
__global__ void YoloPassThruForward(const Dtype* input, int size, int channels, int rows, 
        int cols, Dtype* output) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    int curBatch = idx / channels;
    int curChannel = idx % channels;

    int stride = 2;

    int topChannels = channels / (stride * stride);
    int topRows = rows * stride;
    int topCols = cols * stride;

    for (int h = 0; h < cols; h++) {
        for (int w = 0; w < rows; w++) {
            int bottomIndex = w + rows * (h + cols * (curChannel + channels * curBatch));
            int c2 = curChannel % topChannels;
            int offset = curChannel / topChannels;
            int w2 = w * stride + offset % stride;
            int h2 = h * stride + offset / stride;
            int topIndex = w2 + topRows * (h2 + topCols * (c2 + topChannels * curBatch));
            output[bottomIndex] = input[topIndex];
        }
    }
}

template <typename Dtype>
__global__ void YoloPassThruBackward(const Dtype* outputGrad, int size, int channels, 
        int rows, int cols, Dtype* inputGrad) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    int curBatch = idx / channels;
    int curChannel = idx % channels;

    int stride = 2;

    int topChannels = channels / (stride * stride);
    int topRows = rows * stride;
    int topCols = cols * stride;

    for (int h = 0; h < cols; h++) {
        for (int w = 0; w < rows; w++) {
            int bottomIndex = w + rows * (h + cols * (curChannel + channels * curBatch));
            int c2 = curChannel % topChannels;
            int offset = curChannel / topChannels;
            int w2 = w * stride + offset % stride;
            int h2 = h * stride + offset / stride;
            int topIndex = w2 + topRows * (h2 + topCols * (c2 + topChannels * curBatch));
            inputGrad[topIndex] = outputGrad[bottomIndex];
        }
    }
}


template <typename Dtype>
void YOLOPassThruLayer<Dtype>::reshape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches 	= inputShape[0];
	uint32_t channels 	= inputShape[1];
	uint32_t rows 		= inputShape[2];
	uint32_t cols 		= inputShape[3];

	this->_inputShape[0] = {batches, channels, rows, cols};
	this->_outputData[0]->reshape({batches, channels * 4, rows / 2, cols / 2});
}

template <typename Dtype>
void YOLOPassThruLayer<Dtype>::feedforward() {
	reshape();

    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
    int batchCount = inputShape[0];
    int channelCount = inputShape[1];
    int rowCount = inputShape[2];
    int colCount = inputShape[3];
    int size = batchCount * channelCount;

    const Dtype *inputData = this->_inputData[0]->device_data();
    Dtype *outputData = this->_outputData[0]->mutable_device_data();

    YoloPassThruForward<Dtype><<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        inputData, size, channelCount, rowCount, colCount, outputData);
}

template <typename Dtype>
void YOLOPassThruLayer<Dtype>::backpropagation() {
    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
    int batchCount = inputShape[0];
    int channelCount = inputShape[1];
    int rowCount = inputShape[2];
    int colCount = inputShape[3];

    int size = batchCount * channelCount;

    const Dtype *outputGrad = this->_outputData[0]->device_grad();
    Dtype *inputGrad = this->_inputData[0]->mutable_device_grad();

    YoloPassThruBackward<Dtype><<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        outputGrad, size, channelCount, rowCount, colCount, inputGrad);
}

template void YOLOPassThruLayer<float>::reshape();
template void YOLOPassThruLayer<float>::feedforward();
template void YOLOPassThruLayer<float>::backpropagation();

/****************************************************************************
 * layer callback functions
 ****************************************************************************/
template<typename Dtype>
void* YOLOPassThruLayer<Dtype>::initLayer() {
	YOLOPassThruLayer* layer = NULL;
	SNEW(layer, YOLOPassThruLayer<Dtype>);
	SASSUME0(layer != NULL);
    return (void*)layer;
}

template<typename Dtype>
void YOLOPassThruLayer<Dtype>::destroyLayer(void* instancePtr) {
    YOLOPassThruLayer<Dtype>* layer = (YOLOPassThruLayer<Dtype>*)instancePtr;
    SDELETE(layer);
}

template<typename Dtype>
void YOLOPassThruLayer<Dtype>::setInOutTensor(void* instancePtr, void* tensorPtr,
    bool isInput, int index) {
    SASSERT0(index == 0);

    YOLOPassThruLayer<Dtype>* layer = (YOLOPassThruLayer<Dtype>*)instancePtr;

    if (isInput) {
        SASSERT0(layer->_inputData.size() == 0);
        layer->_inputData.push_back((Data<Dtype>*)tensorPtr);
    } else {
        SASSERT0(layer->_outputData.size() == 0);
        layer->_outputData.push_back((Data<Dtype>*)tensorPtr);
    }
}

template<typename Dtype>
bool YOLOPassThruLayer<Dtype>::allocLayerTensors(void* instancePtr) {
    YOLOPassThruLayer<Dtype>* layer = (YOLOPassThruLayer<Dtype>*)instancePtr;
    layer->reshape();
    return true;
}

template<typename Dtype>
void YOLOPassThruLayer<Dtype>::forwardTensor(void* instancePtr, int miniBatchIdx) {
	YOLOPassThruLayer<Dtype>* layer = (YOLOPassThruLayer<Dtype>*)instancePtr;
	layer->feedforward();
}

template<typename Dtype>
void YOLOPassThruLayer<Dtype>::backwardTensor(void* instancePtr) {
	YOLOPassThruLayer<Dtype>* layer = (YOLOPassThruLayer<Dtype>*)instancePtr;
	layer->backpropagation();
}

template<typename Dtype>
void YOLOPassThruLayer<Dtype>::learnTensor(void* instancePtr) {
    SASSERT0(false);
}

template void* YOLOPassThruLayer<float>::initLayer();
template void YOLOPassThruLayer<float>::destroyLayer(void* instancePtr);
template void YOLOPassThruLayer<float>::setInOutTensor(void* instancePtr, void* tensorPtr,
    bool isInput, int index);
template bool YOLOPassThruLayer<float>::allocLayerTensors(void* instancePtr);
template void YOLOPassThruLayer<float>::forwardTensor(void* instancePtr, int miniBatchIdx);
template void YOLOPassThruLayer<float>::backwardTensor(void* instancePtr);
template void YOLOPassThruLayer<float>::learnTensor(void* instancePtr);


