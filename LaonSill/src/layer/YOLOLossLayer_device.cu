#include "hip/hip_runtime.h"
/**
 * @file YOLOLossLayer_device.cu
 * @date 2017-04-21
 * @author moonhoen lee
 * @brief 
 * @details
 */

#include "hip/hip_runtime.h"

#include "YOLOLossLayer.h"
#include "Network.h"
#include "SysLog.h"
#include "StdOutLog.h"
#include "PropMgmt.h"
#include "MemoryMgmt.h"

using namespace std;

#define YOLOLOSSLAYER_LOG         1

#define EPSILON                 0.000001

template <typename Dtype>
__global__ void YoloBackward(const Dtype* input, const Dtype* input2,
    int size, Dtype noobjVal, Dtype coordVal, Dtype objVal, Dtype classVal, Dtype* grad) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= size)
		return;

    // NOTE: 아래 코드는 필요 없습니다. forward에서 backward까지 진행하기 때문입니다.
#if 0
    Dtype labelClass = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 6];

    int labelClassInt = (int)(labelClass + EPSILON);

    if (labelClassInt == 0) {
        for (int i = 0; i < YOLO_GRID_ELEM_COUNT; i++) {
            grad[idx * YOLO_GRID_ELEM_COUNT + i] = 0.0;
        }

        for (int j = 0; j < YOLO_ANCHOR_BOX_COUNT; j++) {
            int confidenceIndex = 
                idx * YOLO_GRID_ELEM_COUNT + j * YOLO_ELEM_COUNT_PER_ANCHORBOX + 4;
            grad[confidenceIndex] = noobjVal * (2.0) * input[confidenceIndex];
        }

        return;
    }

    Dtype x = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 2];
    Dtype y = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 3];
    Dtype w = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 4];
    Dtype h = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 5];

    // backward boxes & classes
    for (int i = 0; i < YOLO_ANCHOR_BOX_COUNT; i++) {
        int boxBaseIndex = idx * YOLO_GRID_ELEM_COUNT + i * YOLO_ELEM_COUNT_PER_ANCHORBOX;

        grad[boxBaseIndex + 0] = coordVal * (2.0) * (input[boxBaseIndex + 0] - x);
        grad[boxBaseIndex + 1] = coordVal * (2.0) * (input[boxBaseIndex + 1] - y);
        grad[boxBaseIndex + 2] = 
            coordVal * (sqrtf(input[boxBaseIndex + 2] + EPSILON) - sqrtf(w + EPSILON))
                    / sqrtf(input[boxBaseIndex + 2] + EPSILON);
        grad[boxBaseIndex + 3] = 
            coordVal * (sqrtf(input[boxBaseIndex + 3] + EPSILON) - sqrtf(h + EPSILON))
                    / sqrtf(input[boxBaseIndex + 3] + EPSILON);
        grad[boxBaseIndex + 4] = objVal * (2.0) * (input[boxBaseIndex + 4] - 1.0);

        for (int j = 0; j < YOLO_CLASS_COUNT; j++) {
            if (j == labelClassInt - 1) {
                grad[boxBaseIndex + 5 + j] =
                    classVal * (2.0) * (input[boxBaseIndex + 5 + j] - 1.0);
            } else {
                grad[boxBaseIndex + 5 + j] = 
                    classVal * (2.0) * (input[boxBaseIndex + 5 + j]);
            }
        }
    }
#endif
}

// YOLO forward
// 소스가 너무 길다.. 정리해야 할꺼 같다.. 그런데 지금하기는 귀찮다.. ㅜㅜ
template <typename Dtype>
__global__ void YoloForward(const Dtype* input, const Dtype* input2, int size, 
    Dtype noobjVal, Dtype coordVal, Dtype objVal, Dtype classVal, Dtype* output, 
    Dtype* inputGrad) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    Dtype labelClass = input2[idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 6];  // 첫번째 label

    int labelClassInt = (int)(labelClass + EPSILON);

    if (labelClassInt == 0) {

        output[idx] = 0.0;

        for (int i = 0; i < YOLO_ANCHOR_BOX_COUNT; i++) {
            int confidenceIndex = 
                idx * YOLO_GRID_ELEM_COUNT + i * YOLO_ELEM_COUNT_PER_ANCHORBOX + 4;
            int boxBaseIndex = idx * YOLO_GRID_ELEM_COUNT + i * 
                YOLO_ELEM_COUNT_PER_ANCHORBOX;
            Dtype c1 = input[confidenceIndex];
            output[idx] = output[idx] + noobjVal * (c1 - 0.0) * (c1 - 0.0);

            for (int j = 0 ; j < YOLO_ELEM_COUNT_PER_ANCHORBOX; j++) {
                inputGrad[boxBaseIndex + j] = 0.0;
            }

            inputGrad[confidenceIndex] = 2 * noobjVal * (c1 - 0.0);
        }

        return;
    }

    for (int t = 0; t < YOLOINPUT_GTCOUNT_PER_GRID; t++) {
        // ground truth Box
        int labelBaseIndex = idx * YOLOINPUT_ELEMCOUNT_PER_GRID + 
            t * YOLOINPUT_ELEMCOUNT_PER_GT;
        labelClass = input2[labelBaseIndex + 6];
        labelClassInt = (int)(labelClass + EPSILON);

        if (labelClassInt == 0)
            break;

        Dtype x = input2[labelBaseIndex + 2];
        Dtype y = input2[labelBaseIndex + 3];
        Dtype w = input2[labelBaseIndex + 4];
        Dtype h = input2[labelBaseIndex + 5];

        // anchor boxes
        int bestBoxIndex = 0;
        Dtype bestBoxIOU = 0.0; 

        Dtype box_iou[5];

        for (int i = 0; i < YOLO_ANCHOR_BOX_COUNT; i++) {
            int boxBaseIndex = idx * YOLO_GRID_ELEM_COUNT + i * YOLO_ELEM_COUNT_PER_ANCHORBOX;

            Dtype x1 = input[boxBaseIndex + 0];
            Dtype y1 = input[boxBaseIndex + 1];
            Dtype w1 = input[boxBaseIndex + 2];
            Dtype h1 = input[boxBaseIndex + 3];

            // calc box iou
            Dtype left = max(x1 - w1 / 2.0, x - w / 2.0);
            Dtype right = min(x1 + w1 / 2.0, x + w / 2.0);
            Dtype top = max(y1 - h1 / 2.0, y - h / 2.0);
            Dtype bottom = min(y1 + h1 / 2.0, y + h / 2.0);
            Dtype ov_w = right - left;
            Dtype ov_h = bottom - top;

            Dtype b_inter;
            if (ov_w <= 0 || ov_h <= 0)
                b_inter = 0.0;
            else
                b_inter = ov_w * ov_h;
       
            Dtype b_union;
            b_union = w1 * h1 + w * h - b_inter;
            box_iou[i] = b_inter / b_union;

            if (i == 0) {
                bestBoxIndex = 0;
                bestBoxIOU = box_iou[i];
            } else {
                if (bestBoxIOU < box_iou[i]) {
                    bestBoxIndex = i;
                    bestBoxIOU = box_iou[i];
                }
            }
        }

        // forward boxes & classes
        output[idx] = 0.0;
        for (int i = 0; i < YOLO_ANCHOR_BOX_COUNT; i++) {
            if (bestBoxIndex != i)
                continue;

            int boxBaseIndex = idx * YOLO_GRID_ELEM_COUNT + i * YOLO_ELEM_COUNT_PER_ANCHORBOX;
            Dtype x1 = input[boxBaseIndex + 0];
            Dtype y1 = input[boxBaseIndex + 1];
            Dtype w1 = input[boxBaseIndex + 2];
            Dtype h1 = input[boxBaseIndex + 3];
            Dtype c1 = input[boxBaseIndex + 4];

            inputGrad[boxBaseIndex + 0] = coordVal * (x1 - x) * 2.0;
            inputGrad[boxBaseIndex + 1] = coordVal * (y1 - y) * 2.0;
            inputGrad[boxBaseIndex + 2] = coordVal * 
                (sqrtf(w1 + EPSILON) - sqrt(w + EPSILON)) / sqrtf(w1 + EPSILON);
            inputGrad[boxBaseIndex + 3] = coordVal * 
                (sqrtf(h1 + EPSILON) - sqrt(h + EPSILON)) / sqrtf(h1 + EPSILON);
            inputGrad[boxBaseIndex + 4] = coordVal * (c1 - box_iou[i]) * 2.0;

            for (int j = 0; j < YOLO_CLASS_COUNT; j++) {
                if (j == labelClassInt - 1) {
                    inputGrad[boxBaseIndex + 5 + j] = 
                        classVal * (input[boxBaseIndex + 5 + j] - 1.0) * 2.0;
                } else {
                    inputGrad[boxBaseIndex + 5 + j] = 
                        classVal * (input[boxBaseIndex + 5 + j] - 0.0) * 2.0;
                }
            }

            output[idx] = output[idx] + coordVal * (x1 - x) * (x1 - x);
            output[idx] = output[idx] + coordVal * (y1 - y) * (y1 - y);

            output[idx] = output[idx] + coordVal *
                (sqrtf(w1 + EPSILON) - sqrtf(w + EPSILON)) *
                (sqrtf(w1 + EPSILON) - sqrtf(w + EPSILON));

            output[idx] = output[idx] + coordVal *
                (sqrtf(h1 + EPSILON) - sqrtf(h + EPSILON)) *
                (sqrtf(h1 + EPSILON) - sqrtf(h + EPSILON));

            output[idx] = output[idx] + objVal * (c1 - box_iou[i]) * (c1 - box_iou[i]);

            for (int j = 0; j < YOLO_CLASS_COUNT; j++) {
                if (j == labelClassInt - 1) {
                    output[idx] = output[idx] + classVal * 
                        (input[boxBaseIndex + 5 + j] - 1.0) * 
                        (input[boxBaseIndex + 5 + j] - 1.0);
                } else {
                    output[idx] = output[idx] + classVal *
                        (input[boxBaseIndex + 5 + j] - 0.0) * 
                        (input[boxBaseIndex + 5 + j] - 0.0);
                }
            }
        }
    }
}

template <typename Dtype>
YOLOLossLayer<Dtype>::YOLOLossLayer() : LossLayer<Dtype>() {
	this->type = Layer<Dtype>::YOLOLoss;
}

template<typename Dtype>
YOLOLossLayer<Dtype>::~YOLOLossLayer() {

}

template <typename Dtype>
void YOLOLossLayer<Dtype>::reshape() {
	if (!Layer<Dtype>::_adjustInputShape()) {
        const uint32_t count = Util::vecCountByAxis(this->_inputShape[0], 1);
        const uint32_t inputDataCount = this->_inputData[0]->getCountByAxis(1);
        assert(count == inputDataCount);
    }

    if (!Layer<Dtype>::_isInputShapeChanged(0))
        return;

    SASSERT0(this->_inputData.size() == 2);

    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	this->_inputShape[0] = inputShape;
	this->_outputData[0]->reshape(this->_inputShape[0]);

    const vector<uint32_t>& inputShape2 = this->_inputData[1]->getShape();
	this->_inputShape[1] = inputShape2;

	STDOUT_COND_LOG(YOLOLOSSLAYER_LOG, 
        "<%s> layer' input-0 has reshaped as: %dx%dx%dx%d\n",
        SLPROP_BASE(name).c_str(), inputShape[0], inputShape[1], inputShape[2], inputShape[3]);
	STDOUT_COND_LOG(YOLOLOSSLAYER_LOG,
	    "<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n", 
        SLPROP_BASE(name).c_str(), inputShape[0], inputShape[1], inputShape[2], inputShape[3]);
}

template <typename Dtype>
void YOLOLossLayer<Dtype>::feedforward() {
	reshape();

    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
    int batchCount = inputShape[0];
    int size = batchCount * YOLO_GRID_COUNT;

    const Dtype *inputData = this->_inputData[0]->device_data();
    const Dtype *inputData2 = this->_inputData[1]->device_data();
    Dtype *outputData = this->_outputData[0]->mutable_device_data();
    Dtype *inputGrad = this->_inputData[0]->mutable_device_grad();

    YoloForward<Dtype><<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        inputData, inputData2, size, (Dtype)SLPROP(YOLOLoss, noobj),
        (Dtype)SLPROP(YOLOLoss, coord), (Dtype)SLPROP(YOLOLoss, obj), 
        (Dtype)SLPROP(YOLOLoss, class), outputData, inputGrad);
}

template <typename Dtype>
void YOLOLossLayer<Dtype>::backpropagation() {
    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
    int batchCount = inputShape[0];
    int size = batchCount * YOLO_GRID_COUNT;

    const Dtype *inputData = this->_inputData[0]->device_data();
    const Dtype *inputData2 = this->_inputData[1]->device_data();
    Dtype *inputGrad = this->_inputData[0]->mutable_device_grad();

    YoloBackward<Dtype><<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        inputData, inputData2, size, (Dtype)SLPROP(YOLOLoss, noobj),
        (Dtype)SLPROP(YOLOLoss, coord), (Dtype)SLPROP(YOLOLoss, obj), 
        (Dtype)SLPROP(YOLOLoss, class), inputGrad);
}

template <typename Dtype>
Dtype YOLOLossLayer<Dtype>::cost() {
    const Dtype* outputData = this->_outputData[0]->host_data();
    Dtype avg = 0.0;

    const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
    int batchCount = inputShape[0];
    int count = YOLO_GRID_COUNT * batchCount;


    for (int i = 0; i < count; i++) {
        avg += outputData[i];
    }
	return avg / (Dtype)batchCount;
}

/****************************************************************************
 * layer callback functions 
 ****************************************************************************/
template<typename Dtype>
void* YOLOLossLayer<Dtype>::initLayer() {
	YOLOLossLayer* layer = NULL;
	SNEW(layer, YOLOLossLayer<Dtype>);
	SASSUME0(layer != NULL);
    return (void*)layer;
}

template<typename Dtype>
void YOLOLossLayer<Dtype>::destroyLayer(void* instancePtr) {
    YOLOLossLayer<Dtype>* layer = (YOLOLossLayer<Dtype>*)instancePtr;
    SDELETE(layer);
}

template<typename Dtype>
void YOLOLossLayer<Dtype>::setInOutTensor(void* instancePtr, void* tensorPtr,
    bool isInput, int index) {

    YOLOLossLayer<Dtype>* layer = (YOLOLossLayer<Dtype>*)instancePtr;

    if (isInput) {
        SASSERT0(index < 2);
        SASSERT0(layer->_inputData.size() == index);
        layer->_inputData.push_back((Data<Dtype>*)tensorPtr);
    } else {
        SASSERT0(index == 0);
        SASSERT0(layer->_outputData.size() == 0);
        layer->_outputData.push_back((Data<Dtype>*)tensorPtr);
    }
}

template<typename Dtype>
bool YOLOLossLayer<Dtype>::allocLayerTensors(void* instancePtr) {
    YOLOLossLayer<Dtype>* layer = (YOLOLossLayer<Dtype>*)instancePtr;
    layer->reshape();
    return true;
}

template<typename Dtype>
void YOLOLossLayer<Dtype>::forwardTensor(void* instancePtr, int miniBatchIdx) {
    YOLOLossLayer<Dtype>* layer = (YOLOLossLayer<Dtype>*)instancePtr;
    layer->feedforward();
}

template<typename Dtype>
void YOLOLossLayer<Dtype>::backwardTensor(void* instancePtr) {
    YOLOLossLayer<Dtype>* layer = (YOLOLossLayer<Dtype>*)instancePtr;
    layer->backpropagation();
}

template<typename Dtype>
void YOLOLossLayer<Dtype>::learnTensor(void* instancePtr) {
    SASSERT0(false);
}

template class YOLOLossLayer<float>;
