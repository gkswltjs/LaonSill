#include "hip/hip_runtime.h"
/**
 * @file Update_device.cu
 * @date 2017-05-30
 * @author moonhoen lee
 * @brief 
 * @details
 */

#include "Update.h"
#include "Cuda.h"

template <typename Dtype>
__global__ void DoNesterov(int size, const Dtype* dx, Dtype* v_prev, Dtype* v, Dtype* x,
    const Dtype mu, const Dtype lr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Nesterov Alogorithm
     *
     * v_prev = v # back this up
     * v = mu * v - learning_rate * dx # velocity update stays the same
     * x += -mu * v_prev + (1 + mu) * v # position update changes form
     *
     */

    v_prev[idx] = v[idx];
    v[idx] = mu * v[idx] - lr * dx[idx];
    x[idx] += (-1.0) * mu * v_prev[idx] + (1 + mu) * v[idx];
}

template <typename Dtype>
__global__ void DoAdagrad(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Adagrad Alogorithm
     *
     * cache += dx**2
     * x += -learning_rate * dx / (sqrt(cache) + eps)
     *
     */

    cache[idx] += dx[idx] * dx[idx];
    x[idx] += (-1.0) * lr * dx[idx] / (sqrtf(cache[idx]) + eps);
}

template <typename Dtype>
__global__ void DoRMSprop(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype dr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * RMSprop
     *
     * cache = decay_rate * cache + (1 - decay_rate) * dx**2
     * x += - learning_rate * dx / (sqrt(cache) + eps)
     *
     */

    cache[idx] = dr * cache[idx] + (1.0 - dr) * dx[idx] * dx[idx];
    x[idx] += (-1.0) * lr * dx[idx] / (sqrtf(cache[idx]) + eps);
}

#define USE_TENSORFLOW_ADAM         0

template <typename Dtype>
__global__ void DoAdam(int size, const Dtype* dx, Dtype* m, Dtype* v, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype beta1, const Dtype beta2,
    const Dtype decayedBeta1, const Dtype decayedBeta2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Adam
     *
     * m = beta1 * m + (1 - beta1) * dx
     * v = beta2 * v + (1 - beta2) * (dx**2)
     * x += -learning_rate * m / (sqrt(v) + eps)
     *
     */
    m[idx] = beta1 * m[idx] + (1.0 - beta1) * dx[idx];
    v[idx] = beta2 * v[idx] + (1.0 - beta2) * dx[idx] * dx[idx];
#if USE_TENSORFLOW_ADAM
    Dtype learningRate = lr * sqrtf(1.0 - decayedBeta2) / (1.0 - decayedBeta1);
    x[idx] += (-1.0) * learningRate * m[idx] / (sqrtf(v[idx]) + eps);
#else
    x[idx] += (-1.0) * lr * m[idx] / (sqrtf(v[idx]) + eps);

#endif
}

template<typename Dtype>
void Update<Dtype>::doNesterov(int size, const Dtype* dx, Dtype* v_prev, Dtype* v, Dtype* x,
    const Dtype mu, const Dtype lr) {
    DoNesterov<<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        size, dx, v_prev, v, x, mu, lr);
}

template<typename Dtype>
void Update<Dtype>::doAdagrad(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps) {
    DoAdagrad<<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        size, dx, cache, x, lr, eps);
}

template<typename Dtype>
void Update<Dtype>::doRMSprop(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype dr) {

    DoRMSprop<<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        size, dx, cache, x, lr, eps, dr); 
}

template<typename Dtype>
void Update<Dtype>::doAdam(int size, const Dtype* dx, Dtype* m, Dtype* v, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype beta1, const Dtype beta2,
    const Dtype decayedBeta1, const Dtype decayedBeta2) {

    DoAdam<<<SOOOA_GET_BLOCKS(size), SOOOA_CUDA_NUM_THREADS>>>(
        size, dx, m, v, x, lr, eps, beta1, beta2, decayedBeta1, decayedBeta2);
}


template void Update<float>::doNesterov(int size, const float* dx, float* v_prev,
        float* v, float* x, const float mu, const float lr);
template void Update<float>::doAdagrad(int size, const float* dx, float* cache,
        float* x, const float lr, const float eps);
template void Update<float>::doRMSprop(int size, const float* dx, float* cache,
        float* x, const float lr, const float eps, const float dr);
template void Update<float>::doAdam(int size, const float* dx, float* m,
        float* v, float* x, const float lr, const float eps, const float beta1,
        const float beta2, const float decayedBeta1, const float decayedBeta2);
